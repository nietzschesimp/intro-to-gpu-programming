#include <iostream>
#include <cstdlib>
#include <chrono>

#include "cuda_kernels.h"
#include "argument_parser.h"


/*
 * Main function
 * @param argc, number of command line args
 * @param argv, 2D character array representing the commands passed via command line.
 */
int main(int argc, char** argv) {
	// Default values
	int array_size = 1024;
	int num_blocks = 1;
	std::string op = "";

	// read command line arguments
	ArgumentParser parser(argc, argv);
	if (parser.exists("-s"))
		array_size = std::atoi(parser.get_option("-s").c_str());
	if (parser.exists("-b"))
		num_blocks = std::atoi(parser.get_option("-b").c_str());
	if (parser.exists("-o"))
		op = parser.get_option("-o");
	else {
		std::cout << "[ERROR]: No operation indicated.\n";
		return EXIT_FAILURE;
	}

	int total_threads = 1 + ((array_size-1)/num_blocks);
	std::cout << "total threads: " << total_threads << std::endl;

	// Declare pointers for GPU based params
	int *input1_host = new int[array_size];
	int *input2_host = new int[array_size];
	int *result_host = new int[array_size];

	// Fill input arrays
	for (int i = 0; i < array_size; i++) {
		input1_host[i] = i;
		input2_host[i] = rand() % (3-0+1) + 0;
	}
	std::cout << "Inputs:\n";
	for (int i = array_size - 10; i < array_size; i++) {
		std::cout << input1_host[i] << "\t" << input2_host[i] << std::endl;
	}

	// Copy data to device
	int *in1, *in2, *res;
	std::chrono::high_resolution_clock::time_point start_w_transfer = std::chrono::high_resolution_clock::now();
	hipMalloc((void**)&in1, sizeof(int)*array_size);
	hipMalloc((void**)&in2, sizeof(int)*array_size);
	hipMalloc((void**)&res, sizeof(int)*array_size);
	hipMemcpy(in1, input1_host, sizeof(int)*array_size, hipMemcpyHostToDevice);
	hipMemcpy(in2, input2_host, sizeof(int)*array_size, hipMemcpyHostToDevice);

	// Take time without transfer
	std::chrono::high_resolution_clock::time_point start_no_transfer = std::chrono::high_resolution_clock::now();

	// Select which kernel
	if (op == "add") {
		add_kernel_i32<<<num_blocks, total_threads>>>(res, in1, in2, array_size);
	}
	else if (op == "sub") {
		sub_kernel_i32<<<num_blocks, total_threads>>>(res, in1, in2, array_size);
	}
	else if (op == "mul") {
		mul_kernel_i32<<<num_blocks, total_threads>>>(res, in1, in2, array_size);
	}
	else if (op == "mod") {
		mod_kernel_i32<<<num_blocks, total_threads>>>(res, in1, in2, array_size);
	}
	
	// Take time without transfer
	std::chrono::high_resolution_clock::time_point end_no_transfer = std::chrono::high_resolution_clock::now();

	// Copy data from device to host
	hipMemcpy(result_host, res, sizeof(int)*array_size, hipMemcpyDeviceToHost );

	// Determine time
	std::chrono::high_resolution_clock::time_point end_w_transfer = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> diff_no_transfer = end_no_transfer - start_no_transfer;
	std::chrono::duration<double> diff_w_transfer = end_w_transfer - start_w_transfer;
	
	// Print result
	std::cout << "Result:\n";
	for (int i = array_size -10; i < array_size; i++) {
		std::cout << result_host[i] << std::endl;
	}

	// Print time
	std::cout << "Time to process without transfer [" << array_size << "] samples (" << 1000*diff_no_transfer.count() << ") ms\n";
	std::cout << "Time to process with transfer [" << array_size << "] samples {" << 1000*diff_w_transfer.count() << "} ms\n";

	// Free memory
	hipFree(in1);
	hipFree(in2);
	hipFree(res);

	return EXIT_SUCCESS;
}
