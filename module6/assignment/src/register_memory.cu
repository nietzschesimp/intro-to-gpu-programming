#include <iostream>
#include <cstdlib>
#include <chrono>

#include "register_memory_kernels.h"
#include "argument_parser.h"


/*
 * Main function
 * @param argc, number of command line args
 * @param argv, 2D character array representing the commands passed via command line.
 */
int main(int argc, char** argv) {
	// Default values
	int array_size = 1024;
	int num_blocks = 1;
	std::string op = "";

	// read command line arguments
	ArgumentParser parser(argc, argv);
	if (parser.exists("-s"))
		array_size = std::atoi(parser.get_option("-s").c_str());
	if (parser.exists("-b"))
		num_blocks = std::atoi(parser.get_option("-b").c_str());
	if (parser.exists("-o"))
		op = parser.get_option("-o");
	else {
		std::cout << "[ERROR]: No operation indicated.\n";
		return EXIT_FAILURE;
	}

	int total_threads = array_size/num_blocks;
	std::cout << "total threads: " << total_threads << std::endl;
	std::cout << "Total samps proc: " << total_threads*num_blocks << std::endl;

	// Declare pointers for GPU based params
	int *input1_host, *input2_host, *result_host;
	hipHostMalloc((void**)&input1_host, sizeof(int)*array_size);
	hipHostMalloc((void**)&input2_host, sizeof(int)*array_size);
	hipHostMalloc((void**)&result_host, sizeof(int)*array_size);

	// Fill input arrays
	for (int i = 0; i < array_size; i++) {
		input1_host[i] = i;
		input2_host[i] = rand() % (3-0+1) + 0;
	}
	std::cout << "Inputs:\n";
	for (int i = array_size - 10; i < array_size; i++) {
		std::cout << input1_host[i] << "\t" << input2_host[i] << std::endl;
	}

	// Transfer data to GPU
	int *in1, *in2, *res;
	hipHostGetDevicePointer(&in1, input1_host, 0);
	hipHostGetDevicePointer(&in2, input2_host, 0);
	hipHostGetDevicePointer(&res, result_host, 0);
	std::chrono::high_resolution_clock::time_point start = std::chrono::high_resolution_clock::now();

	if (op == "add") {
		add_kernel_register_i32<<<num_blocks, total_threads>>>(res, in1, in2, array_size);
	}
	else if (op == "sub") {
		sub_kernel_register_i32<<<num_blocks, total_threads>>>(res, in1, in2, array_size);
	}
	else if (op == "mul") {
		mul_kernel_register_i32<<<num_blocks, total_threads>>>(res, in1, in2, array_size);
	}
	else if (op == "mod") {
		mod_kernel_register_i32<<<num_blocks, total_threads>>>(res, in1, in2, array_size);
	}

	// Take time without transfer
	std::chrono::high_resolution_clock::time_point end_no_transfer = std::chrono::high_resolution_clock::now();

	// Synchonize data between device and host
	hipDeviceSynchronize();

	// Determine time
	std::chrono::high_resolution_clock::time_point end_w_transfer = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> diff_no_transfer = end_no_transfer - start;
	std::chrono::duration<double> diff_w_transfer = end_w_transfer - start;

	// Print result
	std::cout << "Result:\n";
	for (int i = array_size - 10; i < array_size; i++) {
		std::cout << result_host[i] << std::endl;
	}

	// Print times
	std::cout << "Time to process without transfer [" << array_size << "] samples (" << 1000*diff_no_transfer.count() << ") ms\n";
	std::cout << "Time to process with transfer [" << array_size << "] samples {" << 1000*diff_w_transfer.count() << "} ms\n";

	// Free memory
	hipHostFree(input1_host);
	hipHostFree(input2_host);
	hipHostFree(result_host);

	return EXIT_SUCCESS;
}
