#include <iostream>
#include <cstdlib>

#include "shared_mem_cuda_kernels.h"
#include "argument_parser.h"


/*
 * Main function
 * @param argc, number of command line args
 * @param argv, 2D character array representing the commands passed via command line.
 */
int main(int argc, char** argv) {
	// Default values
	int array_size = 1024;
	int num_blocks = 1;
	std::string op = "";

	// read command line arguments
	ArgumentParser parser(argc, argv);
	if (parser.exists("-s"))
		array_size = std::atoi(parser.get_option("-s").c_str());
	if (parser.exists("-b"))
		num_blocks = std::atoi(parser.get_option("-b").c_str());
	if (parser.exists("-o"))
		op = parser.get_option("-o");
	else {
		std::cout << "[ERROR]: No operation indicated.\n";
		return EXIT_FAILURE;
	}

	// Calculate total number of threads
	int total_threads = array_size/num_blocks;
	std::cout << "total threads: " << total_threads << std::endl;
	std::cout << "Total samps proc: " << total_threads*num_blocks << std::endl;

	// Get device propoerties
	hipDeviceProp_t prop; 
  int which_device; 
  hipGetDeviceCount(&which_device); 
  hipGetDeviceProperties(&prop, which_device);

	// Allocate memory in device for variables
  int *device_a, *device_b, *device_result; 
  hipMalloc( ( void**)& device_a, 			array_size * sizeof ( *device_a ) ); 
  hipMalloc( ( void**)& device_b, 			array_size * sizeof ( *device_b ) ); 
  hipMalloc( ( void**)& device_result, array_size * sizeof ( *device_result ) ); 

	// Allocate memory on host
  int *host_a, *host_b, *host_result; 
  hipHostAlloc((void **)&host_a, 			array_size * sizeof(int), hipHostMallocDefault);
  hipHostAlloc((void **)&host_b, 			array_size * sizeof(int), hipHostMallocDefault);
  hipHostAlloc((void **)&host_result, 	array_size * sizeof(int), hipHostMallocDefault);

	// Fill input arrays
	for (int i = 0; i < array_size; i++) {
		host_a[i] = i;
		host_b[i] = rand() % (3-0+1) + 0;
	}

	// Print input variables
	std::cout << "Inputs:\n";
	for (int i = array_size - 10; i < array_size; i++) {
		std::cout << host_a[i] << "\t" << host_b[i] << std::endl;
	}

	// Create streams
  hipStream_t stream; 
  hipStreamCreate(&stream); 

	// Create events for timing
	hipEvent_t start, stop, kernel_start, kernel_stop; 
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventCreate(&kernel_start);
  hipEventCreate(&kernel_stop);

  // Create start event
  hipEventRecord(start);

  // Copy data to device
  hipMemcpyAsync(device_a, host_a, array_size * sizeof ( int ), hipMemcpyHostToDevice, stream); 
  hipMemcpyAsync(device_b, host_b, array_size * sizeof ( int ), hipMemcpyHostToDevice, stream); 

	// Record start of kernel time
  hipEventRecord(kernel_start);

	// Select which kernel
	if (op == "add") {
		add_kernel_shr_i32<<< num_blocks, total_threads, 3*total_threads*sizeof(int), stream >>>(device_result, device_a, device_b, array_size);
	}
	else if (op == "sub") {
		sub_kernel_shr_i32<<< num_blocks, total_threads, 3*total_threads*sizeof(int), stream >>>(device_result, device_a, device_b, array_size);
	}
	else if (op == "mul") {
		mul_kernel_shr_i32<<< num_blocks, total_threads, 3*total_threads*sizeof(int), stream >>>(device_result, device_a, device_b, array_size);
	}
	else if (op == "mod") {
		mod_kernel_shr_i32<<< num_blocks, total_threads, 3*total_threads*sizeof(int), stream >>>(device_result, device_a, device_b, array_size);
	}

	// Record stop of kernel
	hipEventRecord(kernel_stop);

	// Asynchronously copy data to host
  hipMemcpyAsync(host_result, device_result, array_size * sizeof(int), hipMemcpyDeviceToHost, stream);

	// Synchronize stream
  hipStreamSynchronize(stream);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop); 

	// Calculate elapsed time
  float elapsed_time_transfer, elapsed_time_kernel = 0.0f;
  hipEventElapsedTime(&elapsed_time_transfer, start, stop); 
  hipEventElapsedTime(&elapsed_time_kernel, kernel_start, kernel_stop); 

	// Print result
	std::cout << "Result:\n";
	for (int i = array_size - 10; i < array_size; i++) {
		std::cout << host_result[i] << std::endl;
	}

	// Print times
	std::cout << "Time to process without transfer [" << array_size << "] samples (" << elapsed_time_kernel << ") ms\n";
	std::cout << "Time to process with transfer [" << array_size << "] samples {" << elapsed_time_transfer << "} ms\n";

	// Release memory on host
  hipHostFree(host_a); 
  hipHostFree(host_b); 
  hipHostFree(host_result); 

	// Release memory on device
  hipFree(device_a); 
  hipFree(device_b); 
  hipFree(device_result);

	return EXIT_SUCCESS;
}
