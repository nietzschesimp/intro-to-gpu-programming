#include <iostream>
#include <cstdlib>
#include <chrono>

#include "argument_parser.h"
#include "cuda_kernels.h"


__constant__ int gpu_const_mem[16384];

int main(int argc, char** argv) {
	// Default values
	int array_size = 1024;
	int num_blocks = 1;
	std::string op = "";

	// read command line arguments
	ArgumentParser parser(argc, argv);
	if (parser.exists("-s"))
		array_size = std::atoi(parser.get_option("-s").c_str());
	if (parser.exists("-b"))
		num_blocks = std::atoi(parser.get_option("-b").c_str());
	if (parser.exists("-o"))
		op = parser.get_option("-o");
	else {
		std::cout << "[ERROR]: No operation indicated.\n";
		return EXIT_FAILURE;
	}

	// Calculate number of threads
	int total_threads = 1 + ((array_size-1)/num_blocks);
	std::cout << "total threads: " << total_threads << std::endl;

	// Declare pointers for GPU based params
	int *input1_host = new int[array_size];
	int *input2_host = new int[array_size];
	int *result_host = new int[array_size];

	// Fill input arrays
	for (int i = 0; i < array_size; i++) {
		input1_host[i] = i;
		input2_host[i] = rand() % (3-0+1) + 0;
	}
	std::cout << "Inputs:\n";
	for (int i = array_size - 10; i < array_size; i++) {
		std::cout << input1_host[i] << "\t" << input2_host[i] << std::endl;
	}

	// Copy data to device
	int *cont_mem_ptr;
	std::chrono::high_resolution_clock::time_point start_w_transfer = std::chrono::high_resolution_clock::now();
	hipGetSymbolAddress((void **)&cont_mem_ptr, gpu_const_mem);
	hipMemcpy(cont_mem_ptr, input1_host, array_size*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(&cont_mem_ptr[array_size], input2_host, array_size*sizeof(int), hipMemcpyHostToDevice);

	// Take time without transfer
	std::chrono::high_resolution_clock::time_point start_no_transfer = std::chrono::high_resolution_clock::now();

	// Select which kernel
	if (op == "add") {
		add_kernel_i32<<<num_blocks, total_threads>>>(cont_mem_ptr, cont_mem_ptr, &cont_mem_ptr[array_size], array_size);
	}
	else if (op == "sub") {
		sub_kernel_i32<<<num_blocks, total_threads>>>(cont_mem_ptr, cont_mem_ptr, &cont_mem_ptr[array_size], array_size);
	}
	else if (op == "mul") {
		mul_kernel_i32<<<num_blocks, total_threads>>>(cont_mem_ptr, cont_mem_ptr, &cont_mem_ptr[array_size], array_size);
	}
	else if (op == "mod") {
		mod_kernel_i32<<<num_blocks, total_threads>>>(cont_mem_ptr, cont_mem_ptr, &cont_mem_ptr[array_size], array_size);
	}
	
	// Take time without transfer
	std::chrono::high_resolution_clock::time_point end_no_transfer = std::chrono::high_resolution_clock::now();

	// Copy data from device to host
	hipMemcpy(result_host, cont_mem_ptr, sizeof(int)*array_size, hipMemcpyDeviceToHost);

	// Determine time
	std::chrono::high_resolution_clock::time_point end_w_transfer = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> diff_no_transfer = end_no_transfer - start_no_transfer;
	std::chrono::duration<double> diff_w_transfer = end_w_transfer - start_w_transfer;
	
	// Print result
	std::cout << "Result:\n";
	for (int i = array_size -10; i < array_size; i++) {
		std::cout << result_host[i] << std::endl;
	}

	// Print time
	std::cout << "Time to process without transfer [" << array_size << "] samples (" << 1000*diff_no_transfer.count() << ") ms\n";
	std::cout << "Time to process with transfer [" << array_size << "] samples {" << 1000*diff_w_transfer.count() << "} ms\n";

	hipDeviceReset();

	return EXIT_SUCCESS;
}
