
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>


/*
 * CUDA kernel for addition of a valua to an array
 * @param summand1, first array to add
 * @param summand2, constant value to add to all elements
 * @param result, the address to where store the result
 */
__global__ 
void add_const_kernel(char* result, const char* summand1, const char summand2) {
	const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	result[thread_idx] = summand1[thread_idx] + summand2;
}

/*
 * CUDA kernel for subtraction of a constant value from an array
 * @param minuend, first array to subtract from
 * @param subtrahend, constant value to subtract from array
 * @param result, the address to where store the result
 */
__global__ 
void sub_const_kernel(char* result, const char* minuend, const char subtrahend) {
	const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	result[thread_idx] = minuend[thread_idx] - subtrahend;
}


/*
 * Main function
 * @param argc, number of command line args
 * @param argv, 2D character array representing the commands passed via command line.
 */
int main(int argc, char** argv)
{
	// read command line arguments
	int total_threads = (1 << 20);
	int block_size = 256;
	char shift_amnt = 0;

	if (argc >= 2) {
		shift_amnt = (char)atoi(argv[1]);
	}

	if (argc >= 3) {
		total_threads = atoi(argv[2]);
	}

	if (argc >= 4) {
		block_size = atoi(argv[3]);
	}

	int num_blocks = total_threads/block_size;

	// validate command line arguments
	if (total_threads % block_size != 0) {
		++num_blocks;
		total_threads = num_blocks*block_size;
		
		printf("Warning: Total thread count is not evenly divisible by the block size\n");
		printf("The total number of threads will be rounded up to %d\n", total_threads);
	}

	// Fill input arrays
	const char input_host[256] = "Lorem ipsum dolor sit amet, consectetur adipiscing elit, sed do eiusmod tempor incididunt ut labore et dolore magna aliqua.";
	char result_host[256];
	printf("%s\n", input_host);

	// Declare pointers for GPU based params
	char* in1, *res;

	// Copy data to device
	hipMalloc((void**)&in1, sizeof(char)*256);
	hipMalloc((void**)&res, sizeof(char)*256);
	hipMemcpy(in1, input_host, sizeof(char)*256, hipMemcpyHostToDevice);

	// Add offset for Caesar cypher
	add_const_kernel<<<num_blocks, total_threads>>>(res, in1, shift_amnt);

	// Copy data from device to host
	hipMemcpy(result_host, res, sizeof(char)*256, hipMemcpyDeviceToHost);
	
	// Print result
	printf("Encoded: %s\n", result_host);

	// Subtract Caesar cypher
	sub_const_kernel<<<num_blocks, total_threads>>>(in1, res, shift_amnt);

	// Copy results from device to host
	hipMemcpy(result_host, in1, sizeof(char)*256, hipMemcpyDeviceToHost);

	// Print result
	printf("Decoded: %s\n", result_host);

	// Free memory
	hipFree(in1);
	hipFree(res);

	return EXIT_SUCCESS;
}
