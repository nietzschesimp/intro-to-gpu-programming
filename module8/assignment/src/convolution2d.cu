#include "hip/hip_runtime.h"
#include "argument_parser.h"
#include <iostream>
#include <hipfft/hipfft.h>
#include <string>


typedef float2 fc32;


float kernel[9] = {
	1, 1, 1,
	1, 0, 1,
	1, 1, 1
};

__global__ void kernel_multiply_complex_fc32(fc32* out, const fc32* in, const fc32* taps, const unsigned int size) {
    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x ;
    if (i < size) {
      out[i].x = in[i].x * taps[i].x - in[i].y*taps[i].y;
      out[i].y = in[i].x * taps[i].y + in[i].y*taps[i].x;
    }
}

int main(int argc, char** argv) {
	// Default values
	int no_rows = 1024;
	int no_cols = 1024;

	// read command line arguments
	ArgumentParser parser(argc, argv);
	if (parser.exists("-r"))
		no_rows = std::stoi(parser.get_option("-r"));
	if (parser.exists("-c"))
		no_cols = std::stoi(parser.get_option("-c"));

  // Instantiate FFT plan
	hipfftHandle fft_plan, ifft_plan;

	// Input pointers
	float* host_input;
	float* device_input;

	// Output pointers
	float* host_output;
	float* device_output;

	// Kernel taps
	fc32* device_taps;
	fc32* device_transformed;
  
	// Create fft plan
  hipfftPlan2d(&fft_plan, no_rows, no_cols, HIPFFT_R2C);
  hipfftPlan2d(&ifft_plan, no_rows, no_cols, HIPFFT_C2R);
  
  // Allocate data for kernel taps in memory
  hipMalloc((void**)&device_taps, sizeof(fc32)*no_rows*no_cols);
  
  // Allocate data for transformed input in memory
  hipMalloc((void**)&device_transformed, sizeof(fc32)*no_rows*no_cols);

  // Create memory for input data
  hipHostMalloc((void**)&host_input, sizeof(float)*no_rows*no_cols);
  hipHostGetDevicePointer(&device_input, host_input, 0);

  // Create memory pointers for output
  hipHostMalloc((void**)&host_output, sizeof(float)*no_rows*no_cols);
  hipHostGetDevicePointer(&device_output, host_output, 0);
	
	// Populate input array
	for(int ii = 0; ii < no_rows; ii++) {
		for(int jj = 0; jj < no_cols; jj++) {
			*(host_input + ii*no_rows + jj) = 0.0f;
		}
	}
	
	for(int ii=0; ii < 3; ii++) {
		for(int jj=0; jj < 3; jj++) {
			*(host_input + ii*no_rows + jj) = *(kernel + ii*3 + jj)/no_rows/no_cols;
		}
	}

	std::cout << "KERNEL:\n";
	for(int ii=0; ii < 10; ii++) {
		for(int jj=0; jj < 10; jj++) {
			std::cout << *(host_input + no_rows*ii + jj) << ", ";
		}
		std::cout << std::endl;
	}
	hipMemcpy(device_input, host_input, no_rows*no_cols*sizeof(float), hipMemcpyHostToDevice);

  // Transform taps
  hipfftExecR2C(fft_plan, device_input, device_taps);

	// Populate input array
	for(int ii = 0; ii < no_rows; ii++) {
		for(int jj = 0; jj < no_cols; jj++) {
			*(host_input + no_rows*ii + jj) = (float)(rand() % 2);
		}
	}
	hipMemcpy(device_input, host_input, no_rows*no_cols*sizeof(float), hipMemcpyHostToDevice);

	// Create events for timing
	hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Create start event
  hipEventRecord(start);

	// Transform input
  hipfftExecR2C(fft_plan, device_input, device_transformed);

	// Perform complex multiplication
	int no_blocks = std::max(no_rows*no_cols/1024, 1);
	int no_threads = 1024;
	kernel_multiply_complex_fc32<<<no_threads, no_blocks>>>(device_transformed, device_transformed, device_taps, 16*no_rows*no_cols);

	// Inverse transform putput
	hipfftExecC2R(ifft_plan, device_transformed, device_output);

  // Create start event
  hipEventRecord(stop);

	// Wait for all operations in the GPU to finish
	hipMemcpy(host_output, device_output, no_rows*no_cols*sizeof(float), hipMemcpyDeviceToHost);

	std::cout << "INPUT:\n";
	for(int ii=0; ii < 10; ii++) {
		for(int jj=0; jj < 10; jj++) {
			std::cout << *(host_input + no_rows*ii + jj) << ", ";
		}
		std::cout << std::endl;
	}

	std::cout << "OUTPUT:\n";
	for(int ii=0; ii < 10; ii++) {
		for(int jj=0; jj < 10; jj++) {
			std::cout << *(host_output + no_rows*ii + jj) << ", ";
		}
		std::cout << std::endl;
	}

	float elapsed_time = 0.0f;
  hipEventElapsedTime(&elapsed_time, start, stop); 
	std::cout << "Time to process without transfer [" << no_rows*no_cols << "] samples (" << elapsed_time << ") ms\n";

  hipFree(device_transformed);
  hipFree(device_taps);
  hipHostFree(host_output);
  hipHostFree(host_input);
	hipfftDestroy(fft_plan);

	return 0;
}
