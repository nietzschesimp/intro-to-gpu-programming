#include <iostream>
#include <cstdlib>
#include <chrono>

#include "cuda_kernels.h"


/*
 * Main function
 * @param argc, number of command line args
 * @param argv, 2D character array representing the commands passed via command line.
 */
int main(int argc, char** argv) {

	// read command line arguments
	int array_size = 1024*1024*4;
	int num_blocks = 2;
	int op = -1;

	if (argc >= 2) {
		num_blocks = atoi(argv[1]);
	}

	if (argc >= 3) {
		array_size = atoi(argv[2]);
	}

	if (argc >= 4) {
		if (strncmp(argv[3], "add", 3) == 0) {
			std::cout << "Set to add\n";
			op = 0;
		}
		if (strncmp(argv[3], "sub", 3) == 0) {
			std::cout << "Set to sub\n";
			op = 1;
		}
		if (strncmp(argv[3], "mul", 3) == 0) {
			std::cout << "Set to mul\n";
			op = 2;
		}
		if (strncmp(argv[3], "mod", 3) == 0) {
			std::cout << "Set to mod\n";
			op = 3;
		}
	}

	int total_threads = 1 + ((array_size-1)/num_blocks);
	std::cout << "total threads: " << total_threads << std::endl;

	// Declare pointers for GPU based params
	int *input1_host = new int[array_size];
	int *input2_host = new int[array_size];
	int *result_host = new int[array_size];

	// Fill input arrays
	for (int i = 0; i < array_size; i++) {
		input1_host[i] = i;
		input2_host[i] = rand() % (3-0+1) + 0;
	}
	std::cout << "Inputs:\n";
	for (int i = array_size - 10; i < array_size; i++) {
		std::cout << input1_host[i] << "\t" << input2_host[i] << std::endl;
	}

	// Copy data to device
	int *in1, *in2, *res;
	std::chrono::high_resolution_clock::time_point start_w_transfer = std::chrono::high_resolution_clock::now();
	hipMalloc((void**)&in1, sizeof(int)*array_size);
	hipMalloc((void**)&in2, sizeof(int)*array_size);
	hipMalloc((void**)&res, sizeof(int)*array_size);
	hipMemcpy(in1, input1_host, sizeof(int)*array_size, hipMemcpyHostToDevice);
	hipMemcpy(in2, input2_host, sizeof(int)*array_size, hipMemcpyHostToDevice);

	// Take time without transfer
	std::chrono::high_resolution_clock::time_point start_no_transfer = std::chrono::high_resolution_clock::now();

	// Select which kernel
	switch(op) {
		case 0:
			add_kernel_i32<<<num_blocks, total_threads>>>(res, in1, in2, array_size);
			break;
		case 1:
			sub_kernel_i32<<<num_blocks, total_threads>>>(res, in1, in2, array_size);
			break;
		case 2:
			mul_kernel_i32<<<num_blocks, total_threads>>>(res, in1, in2, array_size);
			break;
		case 3:
			mod_kernel_i32<<<num_blocks, total_threads>>>(res, in1, in2, array_size);
			break;
		default:
			std::cout << "ERROR: No operation indicated.\n";
			return EXIT_FAILURE;
	}
	
	// Take time without transfer
	std::chrono::high_resolution_clock::time_point end_no_transfer = std::chrono::high_resolution_clock::now();

	// Copy data from device to host
	hipMemcpy(result_host, res, sizeof(int)*array_size, hipMemcpyDeviceToHost );

	// Determine time
	std::chrono::high_resolution_clock::time_point end_w_transfer = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> diff_no_transfer = end_no_transfer - start_no_transfer;
	std::chrono::duration<double> diff_w_transfer = end_w_transfer - start_w_transfer;
	
	// Print result
	std::cout << "Result:\n";
	for (int i = array_size -10; i < array_size; i++) {
		std::cout << result_host[i] << std::endl;
	}

	// Print time
	std::cout << "Time to process without transfer [" << array_size << "] samples (" << 1000*diff_no_transfer.count() << ") ms\n";
	std::cout << "Time to process with transfer [" << array_size << "] samples {" << 1000*diff_w_transfer.count() << "} ms\n";

	// Free memory
	hipFree(in1);
	hipFree(in2);
	hipFree(res);

	return EXIT_SUCCESS;
}
