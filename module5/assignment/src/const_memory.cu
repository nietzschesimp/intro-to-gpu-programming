#include <iostream>
#include <cstdlib>
#include <chrono>

#include "cuda_kernels.h"


__constant__ int gpu_const_mem[16384];

int main(int argc, char** argv) {

	// read command line arguments
	int array_size = 1024*1024*4;
	int num_blocks = 2;
	int op = -1;

	// Parse command line arguments
	if (argc >= 2) {
		num_blocks = atoi(argv[1]);
	}

	if (argc >= 3) {
		array_size = atoi(argv[2]);
	}

	if (argc >= 4) {
		if (strncmp(argv[3], "add", 3) == 0) {
			std::cout << "Set to add\n";
			op = 0;
		}
		if (strncmp(argv[3], "sub", 3) == 0) {
			std::cout << "Set to sub\n";
			op = 1;
		}
		if (strncmp(argv[3], "mul", 3) == 0) {
			std::cout << "Set to mul\n";
			op = 2;
		}
		if (strncmp(argv[3], "mod", 3) == 0) {
			std::cout << "Set to mod\n";
			op = 3;
		}
	}

	// Calculate number of threads
	int total_threads = 1 + ((array_size-1)/num_blocks);
	std::cout << "total threads: " << total_threads << std::endl;

	// Declare pointers for GPU based params
	int *input1_host = new int[array_size];
	int *input2_host = new int[array_size];
	int *result_host = new int[array_size];

	// Fill input arrays
	for (int i = 0; i < array_size; i++) {
		input1_host[i] = i;
		input2_host[i] = rand() % (3-0+1) + 0;
	}
	std::cout << "Inputs:\n";
	for (int i = array_size - 10; i < array_size; i++) {
		std::cout << input1_host[i] << "\t" << input2_host[i] << std::endl;
	}

	// Copy data to device
	int *cont_mem_ptr;
	std::chrono::high_resolution_clock::time_point start_w_transfer = std::chrono::high_resolution_clock::now();
	hipGetSymbolAddress((void **)&cont_mem_ptr, gpu_const_mem);
	hipMemcpy(cont_mem_ptr, input1_host, array_size*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(&cont_mem_ptr[array_size], input2_host, array_size*sizeof(int), hipMemcpyHostToDevice);

	// Take time without transfer
	std::chrono::high_resolution_clock::time_point start_no_transfer = std::chrono::high_resolution_clock::now();

	// Select which kernel
	switch(op) {
		case 0:
			add_kernel_i32<<<num_blocks, total_threads>>>(cont_mem_ptr, cont_mem_ptr, &cont_mem_ptr[array_size], array_size);
			break;
		case 1:
			sub_kernel_i32<<<num_blocks, total_threads>>>(cont_mem_ptr, cont_mem_ptr, &cont_mem_ptr[array_size], array_size);
			break;
		case 2:
			mul_kernel_i32<<<num_blocks, total_threads>>>(cont_mem_ptr, cont_mem_ptr, &cont_mem_ptr[array_size], array_size);
			break;
		case 3:
			mod_kernel_i32<<<num_blocks, total_threads>>>(cont_mem_ptr, cont_mem_ptr, &cont_mem_ptr[array_size], array_size);
			break;
		default:
			std::cout << "ERROR: No operation indicated.\n";
			return EXIT_FAILURE;
	}
	
	// Take time without transfer
	std::chrono::high_resolution_clock::time_point end_no_transfer = std::chrono::high_resolution_clock::now();

	// Copy data from device to host
	hipMemcpy(result_host, cont_mem_ptr, sizeof(int)*array_size, hipMemcpyDeviceToHost);

	// Determine time
	std::chrono::high_resolution_clock::time_point end_w_transfer = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> diff_no_transfer = end_no_transfer - start_no_transfer;
	std::chrono::duration<double> diff_w_transfer = end_w_transfer - start_w_transfer;
	
	// Print result
	std::cout << "Result:\n";
	for (int i = array_size -10; i < array_size; i++) {
		std::cout << result_host[i] << std::endl;
	}

	// Print time
	std::cout << "Time to process without transfer [" << array_size << "] samples (" << 1000*diff_no_transfer.count() << ") ms\n";
	std::cout << "Time to process with transfer [" << array_size << "] samples {" << 1000*diff_w_transfer.count() << "} ms\n";

	hipDeviceReset();

	return EXIT_SUCCESS;
}
